#include "hip/hip_runtime.h"
//sudo apt update
//!sudo apt install libfftw3-dev
//!nvcc -o CudaEQ prova.cu -lsndfile -lfftw3f -lcufft
//!ncu --kernel-name applyMultiBandGainKernelOptimized ./CudaEQ
/*size_t size = ((n * sizeof(float) + 511) / 512) * 512;
hipMalloc(&d_data, size);*/
#include <iostream>
#include <vector>
#include <cmath>
#include <sndfile.h>
#include <fftw3.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <chrono>
#include <cstring>

#define SAMPLE_RATE 44100
#define BLOCK_SIZE 256
#define ELEMENTS_PER_THREAD 4
#define ALIGNMENT 32  // Allineamento della memoria a 32 byte

// Costanti in memoria costante
__constant__ float d_gains[3];  // [lowGain, midGain, highGain]
__constant__ int d_bandLimits[2];  // [lowEnd, midEnd]

// Kernel ottimizzato
__global__ void applyMultiBandGainKernelOptimized(float* __restrict__ real, float* __restrict__ imag, const int numSamples) {
    __shared__ float sharedReal[BLOCK_SIZE * ELEMENTS_PER_THREAD];
    __shared__ float sharedImag[BLOCK_SIZE * ELEMENTS_PER_THREAD];

    const int tid = threadIdx.x;
    const int blockOffset = blockIdx.x * blockDim.x * ELEMENTS_PER_THREAD;
    const int globalOffset = blockOffset + tid;

    // Caricamento nella memoria condivisa (con accesso coalescente)
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int idx = globalOffset + i * blockDim.x;
        if (idx < numSamples) {
            sharedReal[tid + i * blockDim.x] = real[idx];
            sharedImag[tid + i * blockDim.x] = imag[idx];
        }
    }
    __syncthreads();

    // Applicazione dei guadagni
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int idx = globalOffset + i * blockDim.x;
        if (idx < numSamples) {
            float gain = (idx < d_bandLimits[0]) * d_gains[0] +
                         (idx >= d_bandLimits[0] && idx < d_bandLimits[1]) * d_gains[1] +
                         (idx >= d_bandLimits[1]) * d_gains[2];
            sharedReal[tid + i * blockDim.x] *= gain;
            sharedImag[tid + i * blockDim.x] *= gain;
        }
    }
    //__syncthreads(); non necessariamente necessante 

    // Scrittura nella memoria globale
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int idx = globalOffset + i * blockDim.x;
        if (idx < numSamples) {
            real[idx] = sharedReal[tid + i * blockDim.x];
            imag[idx] = sharedImag[tid + i * blockDim.x];
        }
    }
}

void applyCudaEqualizer(float* real, float* imag, int numSamples, int sampleRate) {
    // Calcolo degli intervalli delle bande
    int bandLimits[2];
    bandLimits[0] = static_cast<int>(300.0f / (static_cast<float>(sampleRate) / numSamples));  // lowEnd
    bandLimits[1] = static_cast<int>(3000.0f / (static_cast<float>(sampleRate) / numSamples)); // midEnd
    
    // Calcolo dei guadagni
    float gains[3] = {
        std::pow(10.0f, -60.0f / 20.0f),  // LOW_GAIN
        std::pow(10.0f, 2.0f / 20.0f),    // MID_GAIN
        std::pow(10.0f, -3.0f / 20.0f)    // HIGH_GAIN
    };
    
    // Copia delle costanti in memoria costante
    hipMemcpyToSymbol(HIP_SYMBOL(d_gains), gains, sizeof(float) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(d_bandLimits), bandLimits, sizeof(int) * 2);
    
    float *d_real, *d_imag;
    
    // Allocazione memoria sulla GPU con allineamento
    hipMalloc(&d_real, numSamples * sizeof(float));
    hipMalloc(&d_imag, numSamples * sizeof(float));
    
    // Copia dei dati dalla CPU alla GPU
    hipMemcpy(d_real, real, numSamples * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_imag, imag, numSamples * sizeof(float), hipMemcpyHostToDevice);
    
    // Configurazione del lancio del kernel
    const int threadsPerBlock = BLOCK_SIZE;
    const int numBlocks = (numSamples + (threadsPerBlock * ELEMENTS_PER_THREAD) - 1) / (threadsPerBlock * ELEMENTS_PER_THREAD);
    // Esecuzione del kernel
    auto start = std::chrono::high_resolution_clock::now();
    applyMultiBandGainKernelOptimized<<<numBlocks, threadsPerBlock>>>(d_real, d_imag, numSamples);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "CUDA execution time Optimized: " 
              << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() 
              << " µs\n";
    
    // Copia dei risultati dalla GPU alla CPU
    hipMemcpy(real, d_real, numSamples * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(imag, d_imag, numSamples * sizeof(float), hipMemcpyDeviceToHost);
    
    // Liberazione della memoria
    hipFree(d_real);
    hipFree(d_imag);
    
    // Stampa delle configurazioni
    printf("\n=== Equalizer Configuration ===\n");
    printf("Sample rate: %d Hz\n", sampleRate);
    printf("Number of samples: %d\n", numSamples);
    printf("Low band range: 0 to %d\n", bandLimits[0]);
    printf("Mid band range: %d to %d\n", bandLimits[0], bandLimits[1]);
    printf("High band range: %d to %d\n", bandLimits[1], numSamples);
    printf("Low band gain: %.2f\n", gains[0]);
    printf("Mid band gain: %.2f\n", gains[1]);
    printf("High band gain: %.2f\n", gains[2]);
    
    printf("\n=== Kernel Configuration ===\n");
    printf("Grid size: %d blocks\n", numBlocks);
    printf("Block size: %d threads per block\n", BLOCK_SIZE);
    printf("Elements per thread: %d\n", ELEMENTS_PER_THREAD);
    printf("Total threads: %d\n", numBlocks * BLOCK_SIZE);
}

int main(int argc, char* argv[]) {
    const char* inputFile = "/content/drive/MyDrive/Colab Notebooks/fullSong1.wav";
    const char* outputFile = "/content/drive/MyDrive/Colab Notebooks/fullSong5.wav";
    SF_INFO sfInfo{};
    SNDFILE* inFile = sf_open(inputFile, SFM_READ, &sfInfo);
    
    if (!inFile) {
        std::cerr << "[ERR] Reading input file" << std::endl;
        return 1;
    }
    
    int numSamples = sfInfo.frames * sfInfo.channels;
    float* real = (float*)std::aligned_alloc(ALIGNMENT, sizeof(float) * numSamples);
    float* imag = (float*)std::aligned_alloc(ALIGNMENT, sizeof(float) * numSamples);
    std::vector<short> buffer(numSamples);
    
    sf_read_short(inFile, buffer.data(), numSamples);
    sf_close(inFile);
    
    for (int i = 0; i < numSamples; ++i) {
        real[i] = static_cast<float>(buffer[i]);
    }
    std::memset(imag, 0, sizeof(float) * numSamples);
    
    fftwf_complex* fftData = (fftwf_complex*)fftwf_malloc(sizeof(fftwf_complex) * numSamples);
    fftwf_plan forwardPlan = fftwf_plan_dft_r2c_1d(numSamples, real, fftData, FFTW_ESTIMATE);
    fftwf_plan inversePlan = fftwf_plan_dft_c2r_1d(numSamples, fftData, real, FFTW_ESTIMATE);
    
    fftwf_execute(forwardPlan);
    
    for (int i = 0; i < numSamples / 2 + 1; ++i) {
        real[i] = fftData[i][0];
        imag[i] = fftData[i][1];
    }
    
    applyCudaEqualizer(real, imag, numSamples / 2, SAMPLE_RATE);
    
    for (int i = 0; i < numSamples / 2 + 1; ++i) {
        fftData[i][0] = real[i];
        fftData[i][1] = imag[i];
    }
    
    fftwf_execute(inversePlan);
    
    float normalFactor = 1.0f / numSamples;
    for (int i = 0; i < numSamples; ++i) {
        buffer[i] = static_cast<short>(std::round(real[i] * normalFactor));
    }
    
    SNDFILE* outFile = sf_open(outputFile, SFM_WRITE, &sfInfo);
    if (!outFile) {
        std::cerr << "[ERR] Writing output file" << std::endl;
        return 1;
    }
    
    sf_write_short(outFile, buffer.data(), numSamples);
    sf_close(outFile);
    
    fftwf_destroy_plan(forwardPlan);
    fftwf_destroy_plan(inversePlan);
    fftwf_free(fftData);
    free(real);
    free(imag);
    
    return 0;
}
/*#include <iostream>
#include <vector>
#include <cmath>
#include <sndfile.h>
#include <fftw3.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <chrono>
#include <cstring>

#define SAMPLE_RATE 44100
#define BLOCK_SIZE 256
#define ELEMENTS_PER_THREAD 2
#define ALIGNMENT 32  // Allineamento della memoria a 32 byte

// Costanti in memoria costante
__constant__ float d_gains[3];  // [lowGain, midGain, highGain]
__constant__ int d_bandLimits[2];  // [lowEnd, midEnd]
struct Element {
    float real, img;
};
// Kernel ottimizzato
__global__ void applyMultiBandGainKernelOptimized(Element* element, const int numSamples) {
    __shared__ float sharedReal[BLOCK_SIZE * ELEMENTS_PER_THREAD];
    __shared__ float sharedImag[BLOCK_SIZE * ELEMENTS_PER_THREAD];

    const int tid = threadIdx.x;
    const int blockOffset = blockIdx.x * blockDim.x * ELEMENTS_PER_THREAD;
    const int globalOffset = blockOffset + tid;

    // Caricamento nella memoria condivisa (con accesso coalescente)
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int idx = globalOffset + i * blockDim.x;
        if (idx < numSamples) {
            sharedReal[tid + i * blockDim.x] = element[idx].real;
            sharedImag[tid + i * blockDim.x] = element[idx].img;
        }
    }
    __syncthreads();

    // Applicazione dei guadagni
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int idx = globalOffset + i * blockDim.x;
        if (idx < numSamples) {
            float gain = (idx < d_bandLimits[0]) * d_gains[0] +
                         (idx >= d_bandLimits[0] && idx < d_bandLimits[1]) * d_gains[1] +
                         (idx >= d_bandLimits[1]) * d_gains[2];
            sharedReal[tid + i * blockDim.x] *= gain;
            sharedImag[tid + i * blockDim.x] *= gain;
        }
    }
    // Scrittura nella memoria globale
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int idx = globalOffset + i * blockDim.x;
        if (idx < numSamples) {
            element[idx].real = sharedReal[tid + i * blockDim.x];
            element[idx].img = sharedImag[tid + i * blockDim.x];
        }
    }
}

void applyCudaEqualizer(float* real, float* imag, int numSamples, int sampleRate) {
    Element *punti = (Element*)malloc((numSamples) * sizeof(Element));
    for( int i=0; i<numSamples; i++){
        punti[i].real = real[i];  
        punti[i].img = imag[i];  
    }
    // Calcolo degli intervalli delle bande
    int bandLimits[2];
    bandLimits[0] = static_cast<int>(300.0f / (static_cast<float>(sampleRate) / numSamples));  // lowEnd
    bandLimits[1] = static_cast<int>(3000.0f / (static_cast<float>(sampleRate) / numSamples)); // midEnd
    
    // Calcolo dei guadagni
    float gains[3] = {
        std::pow(10.0f, -60.0f / 20.0f),  // LOW_GAIN
        std::pow(10.0f, 2.0f / 20.0f),    // MID_GAIN
        std::pow(10.0f, -3.0f / 20.0f)    // HIGH_GAIN
    };
    
    // Copia delle costanti in memoria costante
    hipMemcpyToSymbol(HIP_SYMBOL(d_gains), gains, sizeof(float) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(d_bandLimits), bandLimits, sizeof(int) * 2);
    
    // Element *elementd;
    Element *elementd;
    hipMalloc(&elementd, (numSamples) * sizeof(Element)); //allochiamo la memoria sul device
    hipMemcpy(elementd, punti, (numSamples) * sizeof(Element), hipMemcpyHostToDevice);
    // Configurazione del lancio del kernel
    const int threadsPerBlock = BLOCK_SIZE;
    const int numBlocks = (numSamples + (threadsPerBlock * ELEMENTS_PER_THREAD) - 1) / (threadsPerBlock * ELEMENTS_PER_THREAD);
    // Esecuzione del kernel
    auto start = std::chrono::high_resolution_clock::now();
    applyMultiBandGainKernelOptimized<<<numBlocks, threadsPerBlock>>>(elementd, numSamples);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "CUDA execution time Optimized: " 
              << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() 
              << " µs\n";
    
    // Copia dei risultati dalla GPU alla CPU
    hipMemcpy(punti, elementd, (numSamples) * sizeof(Element), hipMemcpyDeviceToHost);
    // Assegna i valori finali
    for (int i=0; i<numSamples; i++){
      real[i]=punti[i].real;
      imag[i]=punti[i].img;
    }
    // Liberazione della memoria
    hipFree(elementd);
    
    // Stampa delle configurazioni
    printf("\n=== Equalizer Configuration ===\n");
    printf("Sample rate: %d Hz\n", sampleRate);
    printf("Number of samples: %d\n", numSamples);
    printf("Low band range: 0 to %d\n", bandLimits[0]);
    printf("Mid band range: %d to %d\n", bandLimits[0], bandLimits[1]);
    printf("High band range: %d to %d\n", bandLimits[1], numSamples);
    printf("Low band gain: %.2f\n", gains[0]);
    printf("Mid band gain: %.2f\n", gains[1]);
    printf("High band gain: %.2f\n", gains[2]);
    
    printf("\n=== Kernel Configuration ===\n");
    printf("Grid size: %d blocks\n", numBlocks);
    printf("Block size: %d threads per block\n", BLOCK_SIZE);
    printf("Elements per thread: %d\n", ELEMENTS_PER_THREAD);
    printf("Total threads: %d\n", numBlocks * BLOCK_SIZE);
}

int main(int argc, char* argv[]) {
    const char* inputFile = "/content/drive/MyDrive/Colab Notebooks/fullSong1.wav";
    const char* outputFile = "/content/drive/MyDrive/Colab Notebooks/fullSong888.wav";
    SF_INFO sfInfo{};
    SNDFILE* inFile = sf_open(inputFile, SFM_READ, &sfInfo);
    
    if (!inFile) {
        std::cerr << "[ERR] Reading input file" << std::endl;
        return 1;
    }
    
    int numSamples = sfInfo.frames * sfInfo.channels;
    float* real = (float*)std::aligned_alloc(ALIGNMENT, sizeof(float) * numSamples);
    float* imag = (float*)std::aligned_alloc(ALIGNMENT, sizeof(float) * numSamples);
    std::vector<short> buffer(numSamples);
    
    sf_read_short(inFile, buffer.data(), numSamples);
    sf_close(inFile);
    
    for (int i = 0; i < numSamples; ++i) {
        real[i] = static_cast<float>(buffer[i]);
    }
    std::memset(imag, 0, sizeof(float) * numSamples);
    
    fftwf_complex* fftData = (fftwf_complex*)fftwf_malloc(sizeof(fftwf_complex) * numSamples);
    fftwf_plan forwardPlan = fftwf_plan_dft_r2c_1d(numSamples, real, fftData, FFTW_ESTIMATE);
    fftwf_plan inversePlan = fftwf_plan_dft_c2r_1d(numSamples, fftData, real, FFTW_ESTIMATE);
    
    fftwf_execute(forwardPlan);
    
    for (int i = 0; i < numSamples / 2 + 1; ++i) {
        real[i] = fftData[i][0];
        imag[i] = fftData[i][1];
    }
    
    applyCudaEqualizer(real, imag, numSamples / 2, SAMPLE_RATE);
    
    for (int i = 0; i < numSamples / 2 + 1; ++i) {
        fftData[i][0] = real[i];
        fftData[i][1] = imag[i];
    }
    
    fftwf_execute(inversePlan);
    
    float normalFactor = 1.0f / numSamples;
    for (int i = 0; i < numSamples; ++i) {
        buffer[i] = static_cast<short>(std::round(real[i] * normalFactor));
    }
    
    SNDFILE* outFile = sf_open(outputFile, SFM_WRITE, &sfInfo);
    if (!outFile) {
        std::cerr << "[ERR] Writing output file" << std::endl;
        return 1;
    }
    
    sf_write_short(outFile, buffer.data(), numSamples);
    sf_close(outFile);
    
    fftwf_destroy_plan(forwardPlan);
    fftwf_destroy_plan(inversePlan);
    fftwf_free(fftData);
    free(real);
    free(imag);
    
    return 0;
}
*/
