#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <sndfile.h>
#include <fftw3.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <chrono>
#include <cstring>

#define SAMPLE_RATE 44100
#define BLOCK_SIZE 64
#define ELEMENTS_PER_THREAD 2
#define SHAREDSIZE ELEMENTS_PER_THREAD*BLOCK_SIZE
__constant__ float d_gains[3];  // [lowGain, midGain, highGain]
__constant__ int d_bandLimits[2];  // [lowEnd, midEnd]
/*
__global__ void applyMultiBandGainKernel(float* __restrict__ real, float* __restrict__ imag, const int numSamples) {
    __shared__ float sharedReal[SHAREDSIZE];
    __shared__ float sharedImag[SHAREDSIZE];
    
    // Calcola gli indici per accessi coalescenti
    const int tid = threadIdx.x;
    const int warpSize = 32;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    
    // Calcola l'offset base per il blocco
    const int blockOffset = blockIdx.x * blockDim.x * ELEMENTS_PER_THREAD;
    
    // Per ogni elemento da processare per thread
    #pragma unroll
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        // Calcola l'indice globale con accesso coalescente
        const int globalIdx = blockOffset + laneId + (i * warpSize) + (warpId * warpSize * ELEMENTS_PER_THREAD);
        const int sharedIdx = tid + i * blockDim.x;
        
        if (globalIdx < numSamples) {
            // Carica i dati in shared memory con accessi coalescenti
            sharedReal[sharedIdx] = real[globalIdx];
            sharedImag[sharedIdx] = imag[globalIdx];
        }
    }
    
    __syncthreads();
    
    // Ricalcola gli indici per il processing
    #pragma unroll
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int globalIdx = blockOffset + laneId + (i * warpSize) + (warpId * warpSize * ELEMENTS_PER_THREAD);
        const int sharedIdx = tid + i * blockDim.x;
        
        if (globalIdx < numSamples) {
            // Determina il gain in base alla frequenza
            float gain = (globalIdx < d_bandLimits[0]) ? d_gains[0] : 
                        (globalIdx < d_bandLimits[1]) ? d_gains[1] : d_gains[2];
            
            // Scrivi il risultato con accessi coalescenti
            real[globalIdx] = sharedReal[sharedIdx] * gain;
            imag[globalIdx] = sharedImag[sharedIdx] * gain;
        }
    }
}*/

__global__ void applyMultiBandGainKernel(float* __restrict__ real, float* __restrict__ imag, const int numSamples) {
    __shared__ float sharedReal[SHAREDSIZE];
    __shared__ float sharedImag[SHAREDSIZE];
    
    // Calcola gli indici base per accessi coalescenti
    const int tid = threadIdx.x;
    //const int totalThreads = blockDim.x * gridDim.x;
    const int baseIdx = blockIdx.x * blockDim.x + tid;
    
    // Carica i dati in shared memory con accessi coalescenti
    const int Nuovo=baseIdx*ELEMENTS_PER_THREAD;
    #pragma unroll
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int globalIdx = Nuovo + i;
        const int sharedIdx = tid + i * blockDim.x;
        
        if (baseIdx < numSamples) {
            // Accesso coalescente alla memoria globale
            sharedReal[sharedIdx] = real[globalIdx];
            sharedImag[sharedIdx] = imag[globalIdx];
        }
    }
    __syncthreads();
    float gain = (Nuovo < d_bandLimits[0]) ? d_gains[0] : 
                        (Nuovo < d_bandLimits[1]) ? d_gains[1] : d_gains[2];
    // Processa i dati e scrivi il risultato
    #pragma unroll
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
        const int globalIdx = Nuovo + i ;
        const int sharedIdx = tid + i * blockDim.x;
        
        if (baseIdx < numSamples) {
            // Calcola il gain una sola volta per ogni elemento
            // Accesso coalescente alla memoria globale
            real[globalIdx] = sharedReal[sharedIdx] * gain;
            imag[globalIdx] = sharedImag[sharedIdx] * gain;
        }
    }
}

void applyCudaEqualizer(float* real, float* imag, int numSamples, int sampleRate) {
    int bandLimits[2];
    bandLimits[0] = static_cast<int>(300.0f / (static_cast<float>(sampleRate) / numSamples));
    bandLimits[1] = static_cast<int>(3000.0f / (static_cast<float>(sampleRate) / numSamples));
    
    float gains[3] = {
        std::pow(10.0f, -60.0f / 20.0f),
        std::pow(10.0f, 2.0f / 20.0f),
        std::pow(10.0f, -3.0f / 20.0f)
    };
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_gains), gains, sizeof(float) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(d_bandLimits), bandLimits, sizeof(int) * 2);
    
    float *d_real, *d_imag;
    hipMalloc(&d_real, numSamples * sizeof(float));
    hipMalloc(&d_imag, numSamples * sizeof(float));
    
    hipMemcpy(d_real, real, numSamples * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_imag, imag, numSamples * sizeof(float), hipMemcpyHostToDevice);
    
    const int threadsPerBlock = BLOCK_SIZE;
    const int numBlocks = (numSamples + (threadsPerBlock * ELEMENTS_PER_THREAD) - 1) / (threadsPerBlock * ELEMENTS_PER_THREAD);
    
    auto start = std::chrono::high_resolution_clock::now();
    applyMultiBandGainKernel<<<numBlocks, threadsPerBlock>>>(d_real, d_imag, numSamples);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    
    hipMemcpy(real, d_real, numSamples * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(imag, d_imag, numSamples * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_real);
    hipFree(d_imag);
}

int main(int argc, char* argv[]) {
    const char* inputFile = "/content/drive/MyDrive/Colab Notebooks/fullSong1.wav";
    const char* outputFile = "/content/drive/MyDrive/Colab Notebooks/fullSongaaaaaaaaaaaaaaaaaaa.wav";
    SF_INFO sfInfo{};
    SNDFILE* inFile = sf_open(inputFile, SFM_READ, &sfInfo);
    
    if (!inFile) {
        std::cerr << "[ERR] Reading input file" << std::endl;
        return 1;
    }
    
    int numSamples = sfInfo.frames * sfInfo.channels;
    
    // Allocate real and imag arrays in pinned memory
    float *real, *imag;
    hipError_t err = hipHostMalloc(&real, sizeof(float) * numSamples);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate pinned memory for real array: " 
                  << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    err = hipHostMalloc(&imag, sizeof(float) * numSamples);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate pinned memory for imag array: " 
                  << hipGetErrorString(err) << std::endl;
        hipHostFree(real);  // Clean up previously allocated memory
        return 1;
    }
    
    std::vector<short> buffer(numSamples);
    sf_read_short(inFile, buffer.data(), numSamples);
    sf_close(inFile);
    
    for (int i = 0; i < numSamples; ++i) {
        real[i] = static_cast<float>(buffer[i]);
    }
    std::memset(imag, 0, sizeof(float) * numSamples);
    
    fftwf_complex* fftData = (fftwf_complex*)fftwf_malloc(sizeof(fftwf_complex) * numSamples);
    fftwf_plan forwardPlan = fftwf_plan_dft_r2c_1d(numSamples, real, fftData, FFTW_ESTIMATE);
    fftwf_plan inversePlan = fftwf_plan_dft_c2r_1d(numSamples, fftData, real, FFTW_ESTIMATE);
    
    fftwf_execute(forwardPlan);
    
    for (int i = 0; i < numSamples / 2 + 1; ++i) {
        real[i] = fftData[i][0];
        imag[i] = fftData[i][1];
    }
    
    applyCudaEqualizer(real, imag, numSamples / 2, SAMPLE_RATE);
    
    for (int i = 0; i < numSamples / 2 + 1; ++i) {
        fftData[i][0] = real[i];
        fftData[i][1] = imag[i];
    }
    
    fftwf_execute(inversePlan);
    
    float normalFactor = 1.0f / numSamples;
    for (int i = 0; i < numSamples; ++i) {
        buffer[i] = static_cast<short>(std::round(real[i] * normalFactor));
    }
    
    SNDFILE* outFile = sf_open(outputFile, SFM_WRITE, &sfInfo);
    if (!outFile) {
        std::cerr << "[ERR] Writing output file" << std::endl;
        hipHostFree(real);
        hipHostFree(imag);
        return 1;
    }
    
    sf_write_short(outFile, buffer.data(), numSamples);
    sf_close(outFile);
    
    fftwf_destroy_plan(forwardPlan);
    fftwf_destroy_plan(inversePlan);
    fftwf_free(fftData);
    
    hipHostFree(real);
    hipHostFree(imag);
    
    return 0;
}
